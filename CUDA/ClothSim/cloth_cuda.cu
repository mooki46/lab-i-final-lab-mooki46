#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cloth.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include <time.h>

__global__ void simulatePoint(Point* points, Spring* springs, int N, int M, int num_springs, float dt, float g, float g_on, float m) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int num_points = N * M;
	if (idx >= num_points) return;

	Point* p = &points[idx];

	if (p->fixed) return;

	float fx = 0.0f, fy = 0.0f;

	// calculate spring forces and damping
	for (int i = 0; i < num_springs; i++)
	{
		Spring& s = springs[i];

		int row1 = s.p1.first;
		int col1 = s.p1.second;

		int row2 = s.p2.first;
		int col2 = s.p2.second;

		// Get pointers to the connected points
		Point* p1 = &points[row1 * M + col1];
		Point* p2 = &points[row2 * M + col2];

		// Apply forces only if p is one of the points connected by the spring
		if (p == p1 || p == p2) {
			float dx = p2->x - p1->x;
			float dy = p2->y - p1->y;

			float dist = sqrtf(dx * dx + dy * dy);
			float magnitude = s.spring_coeff * (dist - s.rest_length);

			float spring_force_x = (dist != 0.0f) ? (magnitude * dx / dist) : 0.0f;
			float spring_force_y = (dist != 0.0f) ? (magnitude * dy / dist) : 0.0f;

			float damping_force_x = -p1->vx * s.damp_coeff;
			float damping_force_y = -p1->vy * s.damp_coeff;

			if (p == p1) {
				fx += spring_force_x + damping_force_x;
				fy += spring_force_y + damping_force_y;
			}
			else {
				fx -= spring_force_x - damping_force_x;
				fy -= spring_force_y - damping_force_y;
			}
		}
	}

	float gravity_force_y = g_on ? -g * m : 0.0f;

	fy += gravity_force_y;

	// random external force
	hiprandState state;
	hiprand_init(clock() * idx, 0, 0, &state);

	float ext_force_x = hiprand_uniform(&state) * 2.0f - 1.0f;
	float ext_force_y = hiprand_uniform(&state) * 2.0f - 1.0f;

	ext_force_x *= p->ext_m;
	ext_force_y *= p->ext_m;

	fx += ext_force_x;
	fy += ext_force_y;

	p->ax = fx / m;
	p->ay = fy / m;

	float prev_x = p->x;
	float prev_y = p->y;

	p->x += p->vx * dt + 0.5f * p->ax * dt * dt;
	p->y += p->vy * dt + 0.5f * p->ay * dt * dt;

	//floor collision
	if (p->y < -16.0f) {
		p->y = -16.0f;
		p->vy = 0.0f;
	}

	float new_vx = (p->x - prev_x) / dt;
	float new_vy = (p->y - prev_y) / dt;

	if (p->y == -16.0f) {
		p->vx = -new_vy;
		p->vy = -new_vy;
	}
	else {
		p->vx = new_vx;
		p->vy = new_vy;
	}
}

extern "C" void simulateKernel(Point * points, Spring * springs, int N, int M, int num_springs, float dt, float g, bool g_on, float m) {
	//printf("CUDA simulation started...\n");
	int num_points = N * M;

	Point* d_points;
	Spring* d_springs;

	hipError_t cudaStatus;

	cudaStatus = hipMalloc(&d_points, num_points * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMalloc(&d_springs, num_springs * sizeof(Spring));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	

	cudaStatus = hipMemcpy(d_points, points, num_points * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	cudaStatus = hipMemcpy(d_springs, springs, num_springs * sizeof(Spring), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	int blockSize = 512;

	int numBlocks = (num_points + blockSize - 1) / blockSize;

	simulatePoint << <numBlocks, blockSize >> > (d_points, d_springs, N, M, num_springs, dt, g, g_on, m);

	hipDeviceSynchronize();
	hipGetLastError();

	hipMemcpy(points, d_points, num_points * sizeof(Point), hipMemcpyDeviceToHost);

	hipFree(d_points);
	hipFree(d_springs);
}
